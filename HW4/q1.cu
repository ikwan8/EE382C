
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <vector>
#include <sstream>

#define N (2048*2048)
#define THREADS_PER_BLOCK 51

using namespace std;

__global__ void lastDigit(int *a, int *b) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        b[index] = a[index]%10;
}

__global__ void find_maximum_kernel(int *array, int *min, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cache[256];


	float temp = -1.0;
	while(index + offset < n){
	temp = fminf(temp, array[index + offset]);

	offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fminf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

	__syncthreads();
	i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
			*min = fminf(*min, cache[0]);
	atomicExch(mutex, 0);  //unlock
	}
}



int main()
{
        std::ifstream ifs;
        ifs.open("inp.txt", std::ifstream::in);
        std::string temp;
        std::getline(ifs, temp);


        vector<int> vect;

        std::stringstream ss(temp);

        int i;

        while (ss >> i)
        {
                vect.push_back(i);

                if (ss.peek() == ',' || ss.peek() == ' ' )
                        ss.ignore();
        }

        ifs.close();
        int *h_array;
	int *d_array;
	int *h_min;
	int *d_min;
	int *d_mutex;
	
	h_array = (int*)malloc(N*sizeof(int));
	h_min = (int*)malloc(sizeof(int));
	hipMalloc((void**)&d_array, N*sizeof(int));
	hipMalloc((void**)&d_min, sizeof(int));
	hipMalloc((void**)&d_mutex, sizeof(int));
	hipMemset(d_min, 0, sizeof(int));
	hipMemset(d_mutex, 0, sizeof(int));
	//populate array
	for(i = 0; i < vect.size(); i++){
               h_array[i] = vect[i]; 
        }

        int size = vect.size() * sizeof(int);
        

	hipMemcpy(d_array, h_array, N*sizeof(int), hipMemcpyHostToDevice);
	dim3 gridSize = 256;
	dim3 blockSize = 256;
	find_maximum_kernel<<< gridSize, blockSize >>>(d_array, d_min, d_mutex, N);
	
	hipMemcpy(h_min, d_min, sizeof(float), hipMemcpyDeviceToHost);

	ofstream q1a;
	q1a.open ("q1a.txt");
	q1a << *h_min;
	q1a.close();
	






	//part b
	int *d_b_array;
	int *h_array_b_solution;
	int *d_array_b_solution;
	
        //b = (int *)malloc(size);
	
	h_array_b_solution = (int*)malloc(N*sizeof(int));
        hipMalloc((void **)&d_array_b_solution, size);
        hipMalloc((void **)&d_b_array, size);
	
	hipMemcpy(d_b_array, h_array, size, hipMemcpyHostToDevice);
        lastDigit<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_b_array, d_array_b_solution);

        hipMemcpy(h_array_b_solution, d_array_b_solution, size, hipMemcpyDeviceToHost);
        
	ofstream q1b;
	q1b.open ("q1b.txt");
	
	for(i = 0; i < vect.size(); i++){
                printf("%d, ", h_array_b_solution[i]);
        	q1b << h_array_b_solution[i] << ",";
	}
	q1b.close();

	free(h_array);
	free(h_min);
	free(h_array_b_solution);
	hipFree(d_array);
	hipFree(d_min);
	hipFree(d_mutex);
	hipFree(d_b_array);
	hipFree(d_array_b_solution);
        return 0;
}
