
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <vector>
#include <sstream>

#define N (2048*2048)
#define THREADS_PER_BLOCK 51

using namespace std;

__global__ void lastDigit(int *a, int *b) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        b[index] = a[index]%10;
}




int main()
{
        std::ifstream ifs;
        ifs.open("inp.txt", std::ifstream::in);
        std::string temp;
        std::getline(ifs, temp);


        vector<int> vect;

        std::stringstream ss(temp);

        int i;

        while (ss >> i)
        {
                vect.push_back(i);

                if (ss.peek() == ',' || ss.peek() == ' ' )
                        ss.ignore();
        }

        ifs.close();
        int arr[vect.size()];
        std::copy(vect.begin(), vect.end(), arr);
        for(i = 0; i < vect.size(); i++){
                printf("%d, ", arr[i]);
        }

        int size = vect.size() * sizeof(int);
        int *d_b, *b;
        b = (int *)malloc(size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&arr, size);
        hipMemcpy(d_b, arr, size, hipMemcpyHostToDevice);
        lastDigit<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(arr, d_b);

        hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
        
	ofstream q1b;
	q1b.open ("q1b.txt");
	
	for(i = 0; i < vect.size(); i++){
                printf("%d, ", b[i]);
        	q1b << b[i] << ",";
	}
	q1b.close();
        return 0;
}
